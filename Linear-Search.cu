#include <stdio.h>
#include <hip/hip_runtime.h>

struct SearchConfiguration {
    bool searchComplete;
    int searchValue;
};

__global__ void linearSearchKernel(int *arr, struct SearchConfiguration* searchConfiguration, int N) {
    if(!(searchConfiguration->searchComplete)) {
        int idx = blockIdx.x * blockDim.x + threadIdx.x;
        if(idx < N && (searchConfiguration->searchValue == arr[idx])) {
            // set the indicator to 1 (found)
            searchConfiguration->searchComplete = true;
        }
    }
}

__global__ void assignValue(int *indicator, int value) {
    *indicator = value;
}

int main() {

    int N = 10000000;

    // create and fill sample array
    int *arr = new int[N];
    for(int i=0;i<N;i++) {
        arr[i] = i+1;
    }

    // size of array in bytes
    size_t size = N * sizeof(int);

    // allocate memory on gddr
    int *d_arr;
    hipMalloc(&d_arr, size);

    // copy contents from host to device
    hipMemcpy(d_arr, arr, size, hipMemcpyHostToDevice);

    // Host part of SearchConfiguration
    struct SearchConfiguration h_searchConfiguration;
    h_searchConfiguration.searchComplete = false;
    h_searchConfiguration.searchValue = -1;
    // Mention SearchConfiguration struct for device
    struct SearchConfiguration* d_searchConfiguration;

    // allocate space for struct SearchConfiguration on the device
    hipMalloc((void**)&d_searchConfiguration, sizeof(h_searchConfiguration));

    // copy contents of host struct SearchConfiguration to device struct SearchConfiguration
    hipMemcpy(d_searchConfiguration, &h_searchConfiguration, sizeof(h_searchConfiguration), hipMemcpyHostToDevice);

    // kernel configurations
    int numberOfBlocksPerGrid = (N+1023)/1024;
    int numberOfThreadsPerBlock = 1024;

    // launch kernel
    linearSearchKernel<<<numberOfBlocksPerGrid, numberOfThreadsPerBlock>>>(d_arr, d_searchConfiguration, N);

    // synchronize to wait for result
    hipDeviceSynchronize();

    // copy passed struct here
    hipMemcpy(&h_searchConfiguration, d_searchConfiguration, sizeof(h_searchConfiguration), hipMemcpyDeviceToHost);

    // display result
    printf("result = %d", h_searchConfiguration.searchComplete);

    // free device memory
    hipFree(d_arr);
    // free host memory
    delete[] arr;

    return 0;
}